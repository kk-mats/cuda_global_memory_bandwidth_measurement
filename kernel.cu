
#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>
#include <utility>

constexpr int kb=1<<10;
constexpr int bytes=24;
constexpr int size_per_thread=bytes*kb/(512*sizeof(int));

namespace kernel
{
__global__
void global_to_shared(int *global)
{
	__shared__ int shared[bytes*kb/sizeof(int)];
	__syncthreads();

#pragma unroll 12
	for(int i=0; i<size_per_thread; ++i)
	{
		shared[threadIdx.x*size_per_thread+i]=global[threadIdx.x*size_per_thread+i];
	}
}


__global__
void shared_to_global(int *global)
{
	__shared__ int shared[bytes*kb/sizeof(int)];
	__syncthreads();

#pragma unroll 12
	for(int i=0; i<size_per_thread; ++i)
	{
		global[threadIdx.x*size_per_thread+i]=shared[threadIdx.x*size_per_thread+i];
	}
}

}


auto measure_global_shared()
{
	constexpr int repeat=10000;

	float gs_sum=0, sg_sum=0, time;

	int *global;
	hipMalloc((void **)&global, bytes*kb);

	const dim3 grid(1);
	const dim3 block(512);

	for(int i=0; i<repeat; ++i)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		
		hipEventRecord(start);
		kernel::global_to_shared<<<grid, block>>>(global);
		hipEventRecord(stop);
		hipEventElapsedTime(&time, start, stop);
		gs_sum+=time;


		hipEventRecord(start);
		kernel::shared_to_global<<<grid, block>>>(global);
		hipEventRecord(stop);
		hipEventElapsedTime(&time, start, stop);
		sg_sum+=time;

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	hipFree(global);

	return std::make_pair(gs_sum, sg_sum);
}


int main()
{
	std::cout<<"data size[KB], global to shared[ms], shared to global[ms]"<<std::endl;
	const auto time=measure_global_shared();
	std::cout<<bytes<<","<<time.first<<","<<time.second<<std::endl;
}
